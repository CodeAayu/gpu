
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
__global__ void srt(int* a, int n){

int idx=threadIdx.x;

for(int i=0; i<n-1; i++){

	if(i%2==idx%2 and idx+1<n)
	{
		if(a[idx]>a[idx+1]){
			int t=a[idx] ;
			a[idx]=a[idx+1] ;
			a[idx+1]=t ;		
		}
	}
}

}

int main()
{

int n;	scanf("%d",&n);

int *a_h;

a_h=(int*)malloc(n*sizeof(int));

for(int i=0; i<n; i++)
	a_h[i]=rand()%1000;

printf("earlier:\n");

for(int i=0; i<n; i++)
	printf("%d ",a_h[i]);

printf("\n");
	
int *a_d;

hipMalloc( (void**)&a_d, n*sizeof(int) ) ; 
hipMemcpy( a_d , a_h , n*sizeof(int) , hipMemcpyHostToDevice ) ;

dim3 blockdim=n ;
dim3 griddim=1 ;

srt<<<griddim,blockdim>>>(a_d,n) ;

hipMemcpy( a_h , a_d , n*sizeof(int) ,hipMemcpyDeviceToHost ) ; 

printf("sorted\n"); 

printf("after:\n");

for(int i=0; i<n; i++)
	printf("%d ",a_h[i]);

}

