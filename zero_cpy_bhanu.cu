
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
__global__ void matadd(int *d_a,int *d_b,int *d_c, int n){

int idx=threadIdx.x;
if(idx<n)
d_c[idx]=d_a[idx]+d_b[idx];

}


int main(){

int n;
scanf("%d",&n);
hipEvent_t start,stop;
float escap_time;


hipEventCreate(&start);
hipEventCreate(&stop);

hipEventRecord(start,0);




int *h_a,*h_b,*h_c;

hipHostAlloc((void**)&h_a,20*n*sizeof(int),hipHostMallocDefault);
hipHostAlloc((void**)&h_b,20*n*sizeof(int),hipHostMallocDefault);
hipHostAlloc((void**)&h_c,20*n*sizeof(int),hipHostMallocDefault);


return 0;

}
