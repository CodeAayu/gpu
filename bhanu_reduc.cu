
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void sum(int *a_d, int n,int* maxsum){
	int strid=n/2;
	//printf("fas");
	int t=threadIdx.x;
	while(strid>=1){
	//printf("af");
		__syncthreads();
		if(t<strid){
			a_d[t]=a_d[t]+a_d[strid+t];
			 // printf("threadid=%d val=%d\n",t,a_d[t]);
		}
		strid/=2;
	}
	maxsum[0]=a_d[0];
}

int main(){
	int n=512;
	int a[n];
	for(int i=0; i<n; i++){
		a[i]=i;
	}
	int *a_d,*maxsum;
	hipMalloc((void**)&a_d,n*sizeof(int));
	hipMalloc((void**)&maxsum,sizeof(int));
	//cudaMalloc((void**)&n_d,sizeof(int));
	hipMemcpy(a_d,a,n*sizeof(int),hipMemcpyHostToDevice);
	//for(int i=0; i<n; i++)
	//printf("%d ",a[i]);
	//printf("\n");
	sum<<<1,n>>>(a_d,n,maxsum);
	int maxi[n];
	int max_val[1];
	hipMemcpy(maxi,a_d,n*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(max_val,maxsum,sizeof(int),hipMemcpyDeviceToHost);
//	for(int i=0; i<n; i++)
	printf("%d ",max_val);
	printf("%d ",maxi[0]);
return 0;
}
