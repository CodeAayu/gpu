#include "hip/hip_runtime.h"

#include<stdio.h>

#define tw 2

__global__ void matmul(int *a,int *b, int *c, int n){
	
	int ix=tw*blockIdx.x+threadIdx.x;
	int iy=tw*blockIdx.y+threadIdx.y;
	int idx=n*iy+ix;
	c[idx]=0;
	for(int k=0; k<n; k++){
		c[idx]+=a[ix*n+k]*b[k*n+iy];
	}
}

int main(){
	int n;
	scanf("%d",&n);
	int *a;
	int *b;
	int *c;
	
for(int i=0; i<n; i++){
		for(int j=0; j<n; j++){

			scanf("%d",&a[i][j]);
		}	
	}
	for(int i=0; i<n; i++){
		for(int j=0; j<n; j++){
			scanf("%d",&b[i][j]);
		}	
	}
			
	int *da,*db,*dc;
	hipMalloc((void**)&da,n*n*sizeof(int));
	hipMalloc((void**)&db,n*n*sizeof(int));
	hipMalloc((void**)&dc,n*n*sizeof(int));
	
	hipMemcpy(da,a,n*n*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(db,b,n*n*sizeof(int),hipMemcpyHostToDevice);

	dim3 griddim(ceil(n*1.0/tw),ceil(n*1.0/tw),1);
	dim3 blockdim(tw,tw,1);

	matmul<<<(griddim,blockdim)>>>(da,db,dc,n);
	
	hipMemcpy(c,dc,n*n*sizeof(int),hipMemcpyDeviceToHost);

	for(int i=0; i<n; i++){
		for(int j=0; j<n; j++){
			printf("%d ",c[i][j]);
		}	
		printf("\n");
	}
}
