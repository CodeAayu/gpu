
#include <hip/hip_runtime.h>
#include <stdio.h>

#define tw 2

__global__ void matadd(int *a, int *b, int *c,  int n){
	int ix = tw*blockIdx.x +threadIdx.x;
	int iy = tw*blockIdx.y + threadIdx.y;
	int idx = iy*n+ix;
	if(idx<n*n)
		c[idx]=a[idx]+b[idx];
}

int main(void) {
	int n;
	scanf("%d",&n);
	int a[n][n];
	int b[n][n];
	int c[n][n];
	for(int i=0; i<n; i++){
		for(int j=0; j<n; j++){
			scanf("%d",&a[i][j]);
		}
	}
	for(int i=0; i<n; i++){
		for(int j=0; j<n; j++){
			scanf("%d",&b[i][j]);
		}
	}
	
	int *a_d, *b_d, *c_d;
	
	hipMalloc((void **)&a_d, n*n*sizeof(int));
	hipMalloc((void **)&b_d, n*n*sizeof(int));
	hipMalloc((void **)&c_d, n*n*sizeof(int));
	hipMemcpy(a_d, a, n*n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(b_d, b, n*n*sizeof(int), hipMemcpyHostToDevice);
	
	dim3 dimGrid(n/2,n/2,1);
	dim3 dimBlock(tw,tw,1);
	
	matadd<<<dimGrid,dimBlock>>>(a_d,b_d,c_d,n);
	
	hipMemcpy(c,c_d,n*n*sizeof(int),hipMemcpyDeviceToHost);
	
	for(int i=0; i<n; i++)
		printf("%d ",&c[i]);
	printf("\n");
	
	return 0;
}

