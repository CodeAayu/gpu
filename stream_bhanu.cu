
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
__global__ void matadd(int *d_a,int *d_b,int *d_c, int n){

int idx=threadIdx.x;
if(idx<n)
d_c[idx]=d_a[idx]+d_b[idx];

}


int main(){

int n;
scanf("%d",&n);
hipEvent_t start,stop;
float escap_time;


hipEventCreate(&start);
hipEventCreate(&stop);

hipEventRecord(start,0);

hipStream_t stream;
hipStreamCreate(&stream);




int *h_a,*h_b,*h_c;

hipHostAlloc((void**)&h_a,20*n*sizeof(int),hipHostMallocDefault);
hipHostAlloc((void**)&h_b,20*n*sizeof(int),hipHostMallocDefault);
hipHostAlloc((void**)&h_c,20*n*sizeof(int),hipHostMallocDefault);

for(int i=0; i<20*n; i++){
	h_a[i]=i;
	h_b[i]=i+1;
}

int *d_a,*d_b,*d_c;

hipMalloc((void**)&d_a,n*sizeof(int));
hipMalloc((void**)&d_b,n*sizeof(int));
hipMalloc((void**)&d_c,n*sizeof(int));


for(int i=0; i<20*n; i+=n){
	hipMemcpyAsync(d_a,h_a+i,n*sizeof(int),hipMemcpyHostToDevice,stream);
	hipMemcpyAsync(d_b,h_b+i,n*sizeof(int),hipMemcpyHostToDevice,stream);

matadd<<<1,n,0,stream>>>(d_a,d_b,d_c,n);
	
	hipMemcpyAsync(h_c+i,d_c,n*sizeof(int),hipMemcpyDeviceToHost,stream);
}
hipStreamSynchronize(stream);
hipEventRecord(stop,0);
hipEventSynchronize(stop);
hipEventElapsedTime(&escap_time,start,stop);
printf("Time:%3.1f\n",escap_time);

for(int i=0; i<20*n; i++)
 printf("%d ",h_c[i]);
hipHostFree(h_a);
hipHostFree(h_b);
hipHostFree(h_c);
hipEventDestroy(start);
hipEventDestroy(stop);
hipFree(h_a);
hipFree(h_b);
hipFree(h_c);
hipStreamDestroy(stream);
return 0;

}
