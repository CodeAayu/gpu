
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void add(int* a, int* b, int* c, int n)
{
	int idx = threadIdx.x;
	if(idx<n)
	c[idx] = a[idx]+ b[idx];
}

int main(){
	int n;
	scanf("%d",&n);
	float elapsed_time;
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	
	hipStream_t stream0,stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);
	
	
	int *h_a,*h_b,*h_c;
	hipHostAlloc((void**)&h_a,20*n*sizeof(int),hipHostMallocDefault);
	hipHostAlloc((void**)&h_b,20*n*sizeof(int),hipHostMallocDefault);
	hipHostAlloc((void**)&h_c,20*n*sizeof(int),hipHostMallocDefault);

	for(int i=0; i<20*n; i++){
		h_a[i]=i;
		h_b[i]=i+1;
	}
	
	for(int i=0; i<20*n; i+=n){
		int *d_a,*d_b,*d_c;
		hipMalloc((void**)&d_a,n*sizeof(int));
		hipMalloc((void**)&d_b,n*sizeof(int));
		hipMalloc((void**)&d_c,n*sizeof(int));
		int seg1=n*(7/10);
		int seg2=n-seg1;		

		hipMemcpyAsync(d_a,h_a+i, seg1*sizeof(int),hipMemcpyHostToDevice,stream0);
		hipMemcpyAsync(d_b,h_b+i, seg1*sizeof(int),hipMemcpyHostToDevice,stream0);
		hipMemcpyAsync(d_a,h_a+i+seg1, seg2*sizeof(int),hipMemcpyHostToDevice,stream1);
		hipMemcpyAsync(d_b,h_b+i+seg1, seg2*sizeof(int),hipMemcpyHostToDevice,stream1);

		add<<<1,seg1,0,stream0>>>(d_a,d_b,d_c,seg1);
		add<<<1,seg2,0,stream1>>>(d_a,d_b,d_c,seg2);

		hipMemcpyAsync(h_c+i,d_c,seg1*sizeof(int),hipMemcpyDeviceToHost,stream0);
		hipMemcpyAsync(h_c+i+seg1,d_c,seg2*sizeof(int),hipMemcpyDeviceToHost,stream1);
		
		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_c);
	}
		
		hipStreamSynchronize(stream0);
		hipStreamSynchronize(stream1);
		hipEventRecord(stop,0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed_time,start,stop);
		printf("Time:%3.1f\n",elapsed_time);
		
		for(int i=0; i<20*n; i++)
			printf("%d ",h_c[i]);
		hipHostFree(h_a);
		hipHostFree(h_b);
		hipHostFree(h_c);
		
		hipEventDestroy(stop);
		hipEventDestroy(start);
		
		
		hipStreamDestroy(stream0);
		hipStreamDestroy(stream1);
		return 0;
}

