
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void vecadd(float *a, float *b, float *c, int n)
{
	int i= threadIdx.x + blockDim.x*blockIdx.x;
	if(i<n)
		c[i] = a[i]+b[i];
}


int main(){
	int n;
	scanf("%d",&n);
	int a[n],b[n];
	for(int i=0; i<n; i++)
		scanf("%d",&a[i]);
	for(int i=0;i<n; i++)
		scanf("%d",&b[i]);
	int c[n];
	float *da,*db,*dc;
	int size = n*sizeof(float);
	hipMalloc((void **) &da,size);
	hipMalloc((void **) &db,size);
	hipMalloc((void **) &dc,size);

		
	
	hipMemcpy(da,a,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(db,b,sizeof(int),hipMemcpyHostToDevice);
//	cudaMemcpy(dc,c,sizeof(int),cudaMemcpyHostToDevice);
//	cudaMemcpy(n,n,sizeof(int),cudaMemcpyHostToDevice);
	
	vecadd<<<ceil(n/32.0),15>>>(da,db,dc,n);

	hipMemcpy(c,dc,sizeof(int),hipMemcpyDeviceToHost);
	
	for(int i=0; i<n; i++)
		printf("%d ",c[i]);
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	
return 0;
	
}

