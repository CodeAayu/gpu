
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void maxi(int *d_a,int n){

int strid=(n/2);
int idx=threadIdx.x;
while(strid>0){
	if(idx<strid)
		if(d_a[idx]<d_a[idx+strid])
			d_a[idx]=d_a[idx+strid];
strid=(strid/2);
}

}

int main(){
int n=512;
int *a;
a=(int*)malloc(n*sizeof(int));
for(int i=0; i<n; i++)
	a[i]=i*i*i;

int *d_a;

hipMalloc((void**)&d_a,n*sizeof(int));

hipMemcpy(d_a,a,n*sizeof(int),hipMemcpyHostToDevice);

dim3 blockD(n,1,1);
dim3 gridD(1,1,1);

maxi<<<gridD,blockD>>>(d_a,n);

int *b;
b=(int*)malloc(n*sizeof(int));
hipMemcpy(b,d_a,n*sizeof(int),hipMemcpyDeviceToHost);
printf("%d\n",b[0]);


}

